#include "hip/hip_runtime.h"
/*! \mainpage KPCA CUDA
 * Kernel function implementation on CUDA<br>
 * Haven't found sufficient eigen library - CULA not free (dense eigensolver), cuSOLVER too complicated for my taste - no snipets/examples available.<br>
 * Generalized Histogram Intersection kernel is used. <br>
 * 
 * Tuned for gtm525m - 2 sms, 1536 threads per sm, 8 blocks per sm - 192 threads per block with minimum of 16 blocks for full occupancy. 
 * 
 */


/**
 * @file kernel.cu
 *
 * @author Daniel Stodulka, dstodu@gmail.com
 *
 * @date 2017
 *
 * @brief Basic KPCA functions, loading data, eigen decomposition
 *
 * Kernel implementation on CUDA, eigen decomposition is not included, one kernel function - Generalized Histogram Intersection kernel.<br>
 * 
 * @see http://crsouza.com/2010/03/17/kernel-functions-for-machine-learning-applications/
 */

#include "hip/hip_runtime.h"
#include ""
#include <hipblas.h>
#include "func.h"
#include <stdio.h>

/**
 * @brief GHI on CUDA kernel
 *
 * Computes kernel on CUDA.<br> 
 * Matrices flattened to 1D<br>
 * Simple grid-stride for loop, over all elements, every thread acts as a single point in computation.
 * Suited for data with low dimensionality.
 *
 * @param r result kernel matrix
 * @param a input data matrix
 * @param size data size
 * @param dim data dimension
 * @param alpha
 * @param beta
 */


__global__ void GHI_Kernel(float *r, const float* a, unsigned int size, unsigned int dim, int alpha, int beta)
{
	int i = (threadIdx.x + blockDim.x*blockIdx.x)*dim;
	
	int gridSize = gridDim.x*blockDim.x*dim;
	while (i < size*dim)
	{
		for (int k = 0; k < size*dim; k=k+dim)
		{
			float sum = 0;
			for (int l = 0; l < dim; l++)
			{
				sum += std::fmin(std::pow(std::abs(a[i+l]), alfa), std::pow(std::abs(a[k+l]), beta));
			}
			r[(i / dim)*size + k / dim] = sum;
		}
		
		i+= gridSize;
	}
}

/**
 * @brief Matrix substraction on CUDA
 *
 * @param r result matrix
 * @param a 
 * @param b 
 * @param size data size
 */

__global__ void MxSubs(float* r, const float* a, const float* b, int size)
{
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	while (i < size*size)
	{
		r[i] = a[i] - b[i];
		i += gridDim.x*blockDim.x;
	}
}

/**
 * @brief Matrix addition on CUDA
 *
 * @param r result matrix
 * @param a 
 * @param b 
 * @param size data size
 */

__global__ void MxAdd(float* r, const float* a, const float* b, int size)
{
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	while (i < size*size)
	{
		r[i] = a[i] + b[i];
		i += gridDim.x*blockDim.x;
	}
}


/**
 * @brief Kernel matrix centering
 * Center kernel matrix, to make it zero sum of elements, centered around the origin. <br>
 * Ck = Kernel - OneN*Kernel - KernelOneN - OneNKernelOneN <br>
 * OneN = n*n matrix, where values = 1/n, n = data dimension <br>
 * Uses cublas library for matrix multiplication.
 * 
 * @param r result matrix
 * @param a kernel matrix to center
 * @param size data size
 */

void Center(float* r,float* a, int size);

/**
 * @brief Applies kernel function on data
 *
 * Flattens input data, allocates space on device and applies kernel function.
 *
 * @param data data matrix
 * @param alpha 
 * @param beta 
 * @param r result matrix
 */

void ApplyKernel(std::vector<std::vector<double>> data,double alpha,double beta, float* r);

/**
 * @brief GHI kernel example usage
 *
 *	Loads data, performs GHI kernel and also kernel centering.
 *
 */

int main()
{
	hipDeviceReset();
	std::vector<std::vector<double>> foo = GetData("swiss2.dat");
	float* kernel = new float[foo.size()*foo.size()];
	float* ctKernel = new float[foo.size()*foo.size()];
	ApplyKernel(foo, 2, 2, kernel);
	hipDeviceReset();
	Center(kernel, ctKernel, foo.size());
	hipDeviceReset();
	
    
    return 0;
}

void Center(float* gKernel, float* ctKernel, int kSize)
{
	float* flatOneN = new float[kSize*kSize];
	
	std::fill(flatOneN,flatOneN+kSize*kSize, 1.0 / kSize);

	float* dFlatOneN, *dFlatKernel, *dOneG, *dGOne, *dOneGOne;
	int memSize = sizeof(float)*kSize*kSize;
	const float alpha = 1.0f;
	const float beta = 1.0f;


	hipMalloc((void**) &dFlatKernel,memSize); 
	hipMalloc((void**) &dFlatOneN, memSize);
	hipMemcpy(dFlatKernel, gKernel, memSize, hipMemcpyHostToDevice);
	hipMemcpy(dFlatOneN, flatOneN, memSize, hipMemcpyHostToDevice);

	hipMalloc((void**) &dOneG, memSize);
	hipMalloc((void**) &dGOne, memSize);
	hipMalloc((void**) &dOneGOne, memSize);

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, kSize, kSize, kSize, &alpha, dFlatKernel, kSize, dFlatOneN, kSize, &beta , dOneG,kSize);
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, kSize, kSize, kSize, &alpha, dFlatOneN, kSize, dFlatKernel, kSize, &beta, dGOne, kSize);
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, kSize, kSize, kSize, &alpha, dFlatOneN, kSize, dOneG, kSize, &beta, dOneGOne, kSize);

	int nThreads = 192;
	int nBlocks = (kSize*kSize + nThreads) / nThreads;
	if (nBlocks > 65535)
		nBlocks = 65536;
	if (nBlocks < 16)
		nBlocks = 16;

	MxSubs <<< nBlocks, nThreads >>> (dOneG, dOneG, dGOne, kSize);
	MxSubs <<< nBlocks, nThreads >>> (dFlatKernel, dFlatKernel, dOneG, kSize);
	MxAdd <<< nBlocks, nThreads >>> (dFlatKernel, dFlatKernel, dOneG, kSize);

	hipMemcpy(ctKernel,dFlatKernel,kSize*kSize,hipMemcpyDeviceToHost);
}


void ApplyKernel(std::vector<std::vector<double>> dataArray, double alpha, double beta, float* kernel)
{

	float* flatData;
	
	int kSize = dataArray.size();
	int mSize = dataArray[0].size();
	flatData = new float[kSize*mSize];

	float* ptrK = flatData;
	for (int i = 0; i < kSize; i++)
	{
		std::copy(dataArray[i].begin(), dataArray[i].end(), ptrK);
		ptrK += dataArray[i].size();
	}

	float* dGKernel;
	float* dFlatData;
	
	hipError_t t;
	t = hipMalloc((void**)&dGKernel, sizeof(float)*kSize*kSize);

	if (t != hipSuccess)
	{
		std::cout << "Result device alloc\n" << std::endl;
	}

	t = hipMalloc((void**)&dFlatData,sizeof(float)*kSize*mSize);
	if (t != hipSuccess)
	{
		std::cout << "flatdata device alloc\n" << std::endl;
	}

	t = hipMemcpy(dFlatData, flatData, sizeof(float) * kSize * mSize, hipMemcpyHostToDevice);
	if (t != hipSuccess)
	{
		std::cout << "Flat memcpy\n" << std::endl;
	}

	int nThreads = 192;
	int nBlocks = (kSize*mSize + nThreads)/nThreads;
	if (nBlocks > 65535)
		nBlocks = 65536;
	if (nBlocks < 16)
		nBlocks = 16;

	GHI_Kernel <<< nBlocks, nThreads >>> (dGKernel,dFlatData,kSize,mSize,alpha,beta);

	t = hipGetLastError();
	if (t != hipSuccess)
	{
		std::cout << "GHI failed: " << hipGetErrorString(t);
	}

	t = hipMemcpy(kernel, dGKernel, sizeof(float) * kSize*kSize, hipMemcpyDeviceToHost);
	if (t != hipSuccess)
	{
		std::cout << "Result memcpy failed!" << std::endl;
	}
	delete [] flatData;
}