#include "hip/hip_runtime.h"
/*

KPCA Daniel Stodulka

DISCLAIMER
Haven't found sufficient eigen library - CULA not free (dense eigensolver), cuSOLVER too complicated for my taste - no snipets/examples available, to sum it up no eigen vectors!


KPCA - read data - two examples provided - famous iris data set, variation on swissroll
kernel application - generalized histogram intersection kernel - proved to be quite efficient when dealing with pictures/similar data
kernel center - to achieve zero mean across matrix elements - cublas for matrix mult
all matrices flattened to 1D, 1D grids


tuned for gtm525m - 2 sms, 1536 threads per sm, 8 blocks per sm - 192 threads per block with minimum of 16 blocks for full occupancy 

*/




#include "hip/hip_runtime.h"
#include ""
#include <hipblas.h>
#include "func.h"
#include <stdio.h>

__global__ void GHI_Kernel(float *r, const float* a, unsigned int size, unsigned int dim, int alfa, int beta)
{

	/*
	 flat matrix 
	 suited for data with low dimensionality
	 every thread performs one step in ghi kernel - loops over all other points, sum of min from every "column"
	 num of rows in matrix = num of threads
	*/

	int i = (threadIdx.x + blockDim.x*blockIdx.x)*dim;
	
	int gridSize = gridDim.x*blockDim.x*dim;
	while (i < size*dim)
	{
		for (int k = 0; k < size*dim; k=k+dim)
		{
			float sum = 0;
			for (int l = 0; l < dim; l++)
			{
				sum += std::fmin(std::pow(std::abs(a[i+l]), alfa), std::pow(std::abs(a[k+l]), beta));
			}
			r[(i / dim)*size + k / dim] = sum;
		}
		
		i+= gridSize;
	}
}

__global__ void MxSubs(float* r, const float* a, const float* b, int size) // matrix sub and add
{
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	while (i < size*size)
	{
		r[i] = a[i] - b[i];
		i += gridDim.x*blockDim.x;
	}
}

__global__ void MxAdd(float* r, const float* a, const float* b, int size)
{
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	while (i < size*size)
	{
		r[i] = a[i] + b[i];
		i += gridDim.x*blockDim.x;
	}
}


void Center(float*,float*,int);
void ApplyKernel(std::vector<std::vector<double>>,double,double,float*);

int main()
{
	hipDeviceReset();
	std::vector<std::vector<double>> foo = GetData("swiss2.dat");
	float* kernel = new float[foo.size()*foo.size()];
	float* ctKernel = new float[foo.size()*foo.size()];
	ApplyKernel(foo, 2, 2, kernel);
	hipDeviceReset();
	Center(kernel, ctKernel, foo.size());
	hipDeviceReset();
	
    
    return 0;
}

void Center(float* gKernel, float* ctKernel, int kSize)
{
	/*** centering kernel - zero sum of elements in kernel matrix ***/
	// ctKernel = gKernel - (OneG) - (gOne) + (OneGone) 
	// gKernel/G/g = kernel matrix
	// One = n*n matrix, values = 1/n

	float* flatOneN = new float[kSize*kSize];
	
	std::fill(flatOneN,flatOneN+kSize*kSize, 1.0 / kSize);

	float* dFlatOneN, *dFlatKernel, *dOneG, *dGOne, *dOneGOne;
	int memSize = sizeof(float)*kSize*kSize;
	const float alpha = 1.0f;
	const float beta = 1.0f;


	hipMalloc((void**) &dFlatKernel,memSize); 
	hipMalloc((void**) &dFlatOneN, memSize);
	hipMemcpy(dFlatKernel, gKernel, memSize, hipMemcpyHostToDevice);
	hipMemcpy(dFlatOneN, flatOneN, memSize, hipMemcpyHostToDevice);

	hipMalloc((void**) &dOneG, memSize); // results
	hipMalloc((void**) &dGOne, memSize);
	hipMalloc((void**) &dOneGOne, memSize);

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	//cublas matrix mx
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, kSize, kSize, kSize, &alpha, dFlatKernel, kSize, dFlatOneN, kSize, &beta , dOneG,kSize);
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, kSize, kSize, kSize, &alpha, dFlatOneN, kSize, dFlatKernel, kSize, &beta, dGOne, kSize);
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, kSize, kSize, kSize, &alpha, dFlatOneN, kSize, dOneG, kSize, &beta, dOneGOne, kSize);

	int nThreads = 192;
	int nBlocks = (kSize*kSize + nThreads) / nThreads;
	if (nBlocks > 65535)
		nBlocks = 65536;
	if (nBlocks < 16)
		nBlocks = 16;

	MxSubs <<< nBlocks, nThreads >>> (dOneG, dOneG, dGOne, kSize);
	MxSubs <<< nBlocks, nThreads >>> (dFlatKernel, dFlatKernel, dOneG, kSize);
	MxAdd <<< nBlocks, nThreads >>> (dFlatKernel, dFlatKernel, dOneG, kSize);

	hipMemcpy(ctKernel,dFlatKernel,kSize*kSize,hipMemcpyDeviceToHost);
}

void ApplyKernel(std::vector<std::vector<double>> dataArray, double alpha, double beta, float* kernel)
{

	float* flatData;
	
	int kSize = dataArray.size();
	int mSize = dataArray[0].size();
	flatData = new float[kSize*mSize];

	float* ptrK = flatData;
	for (int i = 0; i < kSize; i++)
	{
		std::copy(dataArray[i].begin(), dataArray[i].end(), ptrK);
		ptrK += dataArray[i].size();
	}

	float* dGKernel;
	float* dFlatData;
	
	hipError_t t;
	t = hipMalloc((void**)&dGKernel, sizeof(float)*kSize*kSize);

	if (t != hipSuccess)
	{
		std::cout << "Result device alloc\n" << std::endl;
	}

	t = hipMalloc((void**)&dFlatData,sizeof(float)*kSize*mSize);
	if (t != hipSuccess)
	{
		std::cout << "flatdata device alloc\n" << std::endl;
	}

	t = hipMemcpy(dFlatData, flatData, sizeof(float) * kSize * mSize, hipMemcpyHostToDevice);
	if (t != hipSuccess)
	{
		std::cout << "Flat memcpy\n" << std::endl;
	}

	int nThreads = 192;
	int nBlocks = (kSize*mSize + nThreads)/nThreads;
	if (nBlocks > 65535)
		nBlocks = 65536;
	if (nBlocks < 16)
		nBlocks = 16;

	GHI_Kernel <<< nBlocks, nThreads >>> (dGKernel,dFlatData,kSize,mSize,alpha,beta);

	t = hipGetLastError();
	if (t != hipSuccess)
	{
		std::cout << "GHI failed: " << hipGetErrorString(t);
	}

	t = hipMemcpy(kernel, dGKernel, sizeof(float) * kSize*kSize, hipMemcpyDeviceToHost);
	if (t != hipSuccess)
	{
		std::cout << "Result memcpy failed!" << std::endl;
	}
	delete [] flatData;
}